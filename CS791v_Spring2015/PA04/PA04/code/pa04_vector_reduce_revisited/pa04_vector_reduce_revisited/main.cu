#include "hip/hip_runtime.h"
/**
 *
 */

#include "hip/hip_runtime.h"
#include ""

#include <numeric>
#include "reduce.hpp"

#include <cstdlib>
#include <cassert>
#include <cstdio>
#include <string>
#include <vector>
#include <fstream>
#include <iostream>
#include <ctime>
#include <map>


/**
 *  A simple structure for easy handling of vector reduction trial results.
 */
typedef struct VectorReduceResult {
  std::string deviceType;
  std::string method;
  unsigned vectorSize;
  float computeTimeSec;
  float totalTimeSec;
  unsigned numBlocks;
  unsigned threadsPerBlock;
  std::string completionStatus;

  VectorReduceResult() {
    deviceType = "";
    method = "";
    computeTimeSec = 0.0;
    totalTimeSec = 0.0;
    numBlocks = 1;
    threadsPerBlock = 1;
    completionStatus = "NOT STARTED";
  };
};


const std::string RELATIVE_REPORT_FOLDER_PATH = "../../../report/";
const std::string CPU_DEVICE = "Intel i5";
const std::string SEQUENTIAL_RUN = "sequential";
const std::string GPU_MULTI_KERNEL_RUN = "gpu with multiple kernel launches";
const std::string GPU_KERNEL_IN_KERNEL_RUN = "gpu kernel launching kernels";
const std::string GPU_WITH_CPU_FINISH = "gpu with cpu final reduction";
const std::string GPU_WITH_THREAD_FENCE = "gpu with thread fencing technique";
const unsigned GPU_MULTI_KERNEL_CODE = 0;
const unsigned GPU_KERNEL_IN_KERNEL_CODE = 1;
const unsigned GPU_WITH_CPU_FINISH_CODE = 2;
const unsigned GPU_WITH_THREAD_FENCE_CODE = 3;


/**
 * Conducts a trial using the single CPU sequential version of vector reduction.
 */
VectorReduceResult
RunSequentialCpuVectorReduceTrial(unsigned vectorSize);

/**
 * Conducts a trial of vector reduce on the GPU, collects the necessary
 * data transfer times, and other pertinent data.
 */
VectorReduceResult
RunGpuVectorReduceTrial(
  unsigned vectorSize,
  unsigned numBlocks,
  unsigned threadsPerBlock,
  unsigned reduceType);

void
GpuMultiKernelReduce(
    int* d_sum,
    int* d_intermediateVector,
    int* d_vector,
    const unsigned vectorSize,
    const unsigned numBlocks,
    const unsigned threadsPerBlock,
    hipDeviceProp_t& gpuProperties);

void
GpuKernelInKernelReduce();

void
GpuWithCpuFinishKernelReduce(
    int* d_sum,
    int* d_intermediateVector,
    int* d_vector,
    const unsigned vectorSize,
    const unsigned numBlocks,
    const unsigned threadsPerBlock,
    hipDeviceProp_t& gpuProperties);

void
GpuThreadFenceReduce();


/**
 * Dynamically allocates a vector with the given parameters and fills the
 * values in so that the results of summing can be checked appropriately.
 */
bool
AllocateTestVectorOnHost(int** vector, const unsigned size);

/**
 * Allocates a test vector on the GPU and returns it by reference.
 * Return value indicates status of the CUDA operation.
 */
hipError_t
AllocateTestVectorMemoryOnGpu(int** device_vector, const unsigned size);

/**
 * My reimplementation of the std::accumulate. I don't know what optimiztiaons
 * std::accumulate may or may not use, so I don't know if it is fine to use
 * it when calculating speed-up.
 */
int
MyAccumulate(const int* vector, unsigned size);

/**
 * Writes the results of all of the trials to a .csv file suited for plotting
 * graphs in Excel.
 */
bool
DumpResultsToExcelDataFile(
  const std::string& filePath,
  const std::string& fileName,
  const std::string& delimeter,
  const std::vector<VectorReduceResult>& results);

/**
 * Writes the results of all of the trials to a .csv file suited for plotting
 * graphs in R.
 */
bool
DumpResultsToRDataFile(
  const std::string& filePath,
  const std::string& fileName,
  const std::string& delimeter,
  const std::vector<VectorReduceResult>& results);

/**
 * Gets relevant GPU properties from the installed device and puts them into
 * a formated string for inclusion as a csv table.
 */
std::string
GetGpuProperties(const int deviceNumber);

void debugIntermediateVectorSum(int* d_arr, int size) {
  int bytes = size * sizeof(int);
  int* p = (int*) malloc(bytes);
  hipMemcpy(p, d_arr, bytes, hipMemcpyDeviceToHost);
  std::cout<<"intermediate sum: "<<std::accumulate(p, p+size,0)<<std::endl;
  free(p);
}

void debugIntermediateVectorArray(int* d_arr, int size) {
  int bytes = size * sizeof(int);
  int* p = (int*) malloc(bytes);
  hipMemcpy(p, d_arr, bytes, hipMemcpyDeviceToHost);
  std::cout<<"values:"<<std::endl;
  unsigned i = 0;
  while (i < size) {
    std::cout<<p[i]<<" ";
    i++;
  }
  std::cout<<std::endl;
  free(p);
}


/**
 * A test harness for collecting data on vector reduction using both the CPU
 * and the GPU.
 */
int
main(const int argc, const char** argv) {
  hipDeviceProp_t gpuProperties;
  hipError_t cudaStatus = hipGetDeviceProperties(&gpuProperties, 0);
  //__cpuid(); //see wikipedia for example


  if (cudaStatus == hipSuccess) {
    std::vector<VectorReduceResult> results;
    unsigned vectorSize;
    unsigned numBlocks;
    unsigned threadsPerBlock;

    for (vectorSize =  1000000; //21 000 000
         vectorSize >= 1000000;
         vectorSize -=  100000) { //1 000 000

      std::cout << "Using vector of " << vectorSize << " values" << std::endl;

      for (double jitter = 0.9; jitter < 1.2; jitter += 0.1) {

        for (numBlocks = gpuProperties.maxGridSize[0];
             numBlocks >= 1;
             numBlocks = ((numBlocks + 1) / 2) - 1) {

          for (threadsPerBlock = gpuProperties.maxThreadsPerBlock;
               threadsPerBlock >= 3; // 1
               threadsPerBlock /= 2) {

            if (numBlocks != 1 || threadsPerBlock != 1) {
              results.push_back(
                RunGpuVectorReduceTrial(
                  vectorSize * jitter,
                  numBlocks,
                  threadsPerBlock,
                  GPU_KERNEL_IN_KERNEL_CODE
                )
              );

              std::cout << ".";
            }
          }
        }

        results.push_back(
          RunSequentialCpuVectorReduceTrial(vectorSize * jitter)
         );
        std::cout << "." << std::endl;
      }
    }

    DumpResultsToExcelDataFile(
      RELATIVE_REPORT_FOLDER_PATH,
      "results.csv",
      ", ",
      results
    );
  } else {
    printf(
      "The GPU doesn't seem to be functioning properly.\r\n"
      "Please check the meaning of error code %i and fix the issue.\r\n",
      cudaStatus
    );
    return 1;
  }

  return 0;
}


VectorReduceResult
RunSequentialCpuVectorReduceTrial(unsigned vectorSize) {
  VectorReduceResult result;
  result.deviceType = CPU_DEVICE;
  result.vectorSize = vectorSize;

  int* testVector = nullptr;
  bool success = AllocateTestVectorOnHost(&testVector, vectorSize);
  if (!success) {
    result.completionStatus = "Failure to allocate host memory";
    return result;
  }

  clock_t computeStart, computeEnd;

  computeStart = clock();

  int sum = MyAccumulate(testVector, vectorSize);

  computeEnd = clock() + 1; // smoothing for really fast runs

  result.computeTimeSec =
    (double) (((double) computeEnd) - (double) computeStart) /
    (double) CLOCKS_PER_SEC;
  result.totalTimeSec = result.computeTimeSec;

  result.completionStatus = "Complete";

  free(testVector); testVector = nullptr;

  return result;
}

VectorReduceResult
RunGpuVectorReduceTrial(
    unsigned vectorSize,
    unsigned numBlocks,
    unsigned threadsPerBlock,
    unsigned reduceType) {

  VectorReduceResult result;
  result.vectorSize = vectorSize;
  result.numBlocks = numBlocks;
  result.threadsPerBlock = threadsPerBlock;

  hipDeviceProp_t gpuProperties;
  hipError_t cudaStatus = hipGetDeviceProperties(&gpuProperties, 0);
  if (cudaStatus != hipSuccess) {
    result.completionStatus = "GPU NOT FOUND!";
    return result;
  }
  result.deviceType = gpuProperties.name;

  int* testVector = nullptr;
  int* d_testVector = nullptr;
  int* intermediateVector = nullptr;
  int* d_intermediateVector = nullptr;
  int* d_sum = nullptr;
  int sum;

  cudaStatus = AllocateTestVectorMemoryOnGpu(&d_testVector, vectorSize);
  if(cudaStatus != hipSuccess) {
    result.completionStatus = "GPU ALLOCATION FAILURE!";
    return result;
  }
  cudaStatus = AllocateTestVectorMemoryOnGpu(
    &d_intermediateVector, numBlocks);
  if(cudaStatus != hipSuccess) {
    hipFree(d_testVector);
    result.completionStatus = "GPU ALLOCATION FAILURE!";
    return result;
  }
  cudaStatus = AllocateTestVectorMemoryOnGpu(&d_sum, 1);
  if(cudaStatus != hipSuccess) {
    hipFree(d_testVector);
    hipFree(d_intermediateVector);
    result.completionStatus = "GPU ALLOCATION FAILURE!";
    return result;
  }
  bool success = AllocateTestVectorOnHost(&testVector, vectorSize);
  if (!success) {
    hipFree(d_testVector);
    hipFree(d_intermediateVector);
    hipFree(d_sum);
    result.completionStatus = "Failure to allocate host memory";
    return result;
  }
  success = AllocateTestVectorOnHost(&intermediateVector, numBlocks);
  if (!success) {
    hipFree(d_testVector);
    hipFree(d_intermediateVector);
    hipFree(d_sum);
    free(testVector);
    result.completionStatus = "Failure to allocate host memory";
    return result;
  }

  hipEvent_t totalStart, computeStart, computeEnd, totalEnd, debug_sync;
  hipEventCreate(&totalStart);
  hipEventCreate(&computeStart);
  hipEventCreate(&computeEnd);
  hipEventCreate(&totalEnd);
  hipEventCreate(&debug_sync);
  hipEventRecord(totalStart, 0);

  hipMemcpy(
    d_testVector,
    testVector,
    vectorSize * sizeof(int),
    hipMemcpyHostToDevice
  );

  hipEventRecord(computeStart, 0);

  switch (reduceType) {
    case GPU_MULTI_KERNEL_CODE:
      result.method = GPU_MULTI_KERNEL_RUN;
      GpuMultiKernelReduce(
        d_sum,
        d_intermediateVector,
        d_testVector,
        vectorSize,
        numBlocks,
        threadsPerBlock,
        gpuProperties
      );
      break;
    case GPU_KERNEL_IN_KERNEL_CODE:
      result.method = GPU_KERNEL_IN_KERNEL_RUN;
      GpuKernelInKernelReduce();
      break;
    case GPU_WITH_CPU_FINISH_CODE:
      result.method = GPU_WITH_CPU_FINISH;
      GpuWithCpuFinishKernelReduce(
        d_sum,
        d_intermediateVector,
        d_testVector,
        vectorSize,
        numBlocks,
        threadsPerBlock,
        gpuProperties
      );
    
      hipEventRecord(computeEnd, 0);
      hipEventSynchronize(computeEnd);

      hipMemcpy(
        intermediateVector, 
        d_intermediateVector,
        numBlocks * sizeof(int),
        hipMemcpyDeviceToHost
      );

      sum = MyAccumulate(intermediateVector, numBlocks);
      break;
    //case GPU_WITH_THREAD_FENCE_CODE:
    //  result.method = GPU_WITH_THREAD_FENCE;
    //  GpuThreadFenceReduce();
    //  break;
    default:
      result.completionStatus = "Invalid method selected";
      return result;
      break;
  }

  hipEventRecord(computeEnd, 0);
  hipEventSynchronize(computeEnd);

  if (reduceType != GPU_WITH_CPU_FINISH_CODE) {
    cudaStatus = hipMemcpy(
      &sum,
      d_sum,
      1 * sizeof(int),
      hipMemcpyDeviceToHost
    );
  }

  if (sum != std::accumulate(testvector, testvector + vectorsize, 0)) {
    std::cout<<"size: "<<vectorsize<<std::endl<<"blocks: "<<numblocks<<std::endl<<"threads: "<<threadsperblock<<std::endl;
    std::cout<<"sum: "<<sum<<std::endl<<"myacc: "<<std::accumulate(testvector, testvector + vectorsize, 0)<<std::endl;
    std::cout<<std::endl<<std::endl;
    debugintermediatevectorsum(d_intermediatevector, numblocks);
    debugintermediatevectorarray(d_intermediatevector, numblocks);
    assert(sum == myaccumulate(testvector, vectorsize));
  }

  hipEventRecord(totalEnd, 0);
  hipEventSynchronize(totalEnd);

  cudaStatus = hipGetLastError();
  if (cudaStatus != hipSuccess) {
    result.completionStatus = "GPU kernel failed to complete";
    result.completionStatus +=
      ": " + std::string(hipGetErrorName(cudaStatus));
    return result;
  } else if (
    sum != std::accumulate(testVector, testVector + vectorSize, 0)) {
    result.completionStatus = "GPU failed to reduce correctly!";
    return result;
  }

  if (reduceType == GPU_WITH_CPU_FINISH_CODE) {
    hipEventElapsedTime(&(result.computeTimeSec), computeStart, computeEnd);
    result.computeTimeSec /= 1000.0;
  }

  hipEventElapsedTime(&(result.totalTimeSec), totalStart, totalEnd);
  result.totalTimeSec /= 1000.0;

  result.completionStatus = "Complete";

  hipFree(d_testVector); d_testVector = nullptr;
  hipFree(d_intermediateVector); d_intermediateVector = nullptr;
  hipFree(d_sum); d_sum = nullptr;
  free(testVector); testVector = nullptr;

  return result;

}

void
GpuMultiKernelReduce(
    int* d_sum,
    int* d_intermediateVector,
    int* d_vector,
    const unsigned vectorSize,
    const unsigned numBlocks,
    const unsigned threadsPerBlock,
    hipDeviceProp_t& gpuProperties) {

  unsigned nBlocks = numBlocks;
  unsigned vSize = vectorSize;
  unsigned sharedBytesNeeded = (threadsPerBlock) * sizeof(int);

  PartialReduceKernel
    <<<nBlocks, threadsPerBlock, sharedBytesNeeded>>>(
    d_intermediateVector,
    d_vector,
    vSize
  );

  while (nBlocks > gpuProperties.maxThreadsPerBlock) {
    vSize = nBlocks;
    nBlocks /= (gpuProperties.maxThreadsPerBlock / 2);

    PartialReduceKernel
      <<<nBlocks, threadsPerBlock, sharedBytesNeeded>>>(
      d_intermediateVector,
      d_intermediateVector,
      vSize
    );
  }

  PartialReduceKernel // TODO: more threads than elements for this call?
    <<<1, nBlocks, nBlocks * sizeof(int)>>>(
    d_sum,
    d_intermediateVector,
    nBlocks
  );
}

void
GpuKernelInKernelReduce(
    int* d_sum,
    int* d_intermediateVector,
    int* d_vector,
    const unsigned vectorSize,
    const unsigned numBlocks,
    const unsigned threadsPerBlock,
    hipDeviceProp_t& gpuProperties) {

  KernelInKernelReduceKernel
  <<<numBlocks, threadsPerBlock, threadsPerBlock * sizeof(int)>>>(
    d_sum,
    d_intermediateVector,
    d_vector,
    vectorSize
  );
}

void
GpuWithCpuFinishKernelReduce(
    int* d_sum,
    int* d_intermediateVector,
    int* d_vector,
    const unsigned vectorSize,
    const unsigned numBlocks,
    const unsigned threadsPerBlock,
    hipDeviceProp_t& gpuProperties) {

  unsigned nBlocks = numBlocks;
  unsigned vSize = vectorSize;
  unsigned sharedBytesNeeded = (threadsPerBlock) * sizeof(int);

  PartialReduceKernel
    <<<nBlocks, threadsPerBlock, sharedBytesNeeded>>>(
    d_intermediateVector,
    d_vector,
    vSize
  );

  // because we are timing different intervals, the cpu reduction will take
  // outside of the function call
  // ... the memcpy and all.
}



void
GpuThreadFenceReduce() {/* TODO: implement */}



bool
AllocateTestVectorOnHost(int** testVector, const unsigned size) {
  unsigned num_bytes = size * sizeof(int);

  *testVector = (int*) malloc(size * sizeof(int));
  if (*testVector == nullptr) {
std::cout<<"failed to allocate "<<num_bytes<<" on host"<<std::endl;
    return false;
  }

  for (unsigned i = 0; i < size; i++) {
    (*testVector)[i] = 1;
  }

  return true;
}

hipError_t
AllocateTestVectorMemoryOnGpu(int** dev_vector, const unsigned size) {
  size_t num_bytes = size * sizeof(int);
  return hipMalloc(dev_vector, num_bytes);
}

int
MyAccumulate(const int* input_vector, unsigned size) {
  int sum = 0;
  for (unsigned i = 0; i < size; ++i) {
    sum += input_vector[i];
  }
  return sum;
}

bool
DumpResultsToExcelDataFile(
  const std::string& file_path,
  const std::string& file_name,
  const std::string& delimeter,
  const std::vector<VectorReduceResult>& results) {

  std::ofstream fout;
  fout.clear();
  fout.open(file_path + file_name);

  if (!fout.good()) {
    puts("Invalid output destination!");
    return false;
  }

  fout << "Device Type" << delimeter
       << "Vector Size" << delimeter
       << "Compute Time (s)" << delimeter
       << "Total Time (s)" << delimeter
       << "Compute Throughput (Flops)" << delimeter
       << "Total Throughput (Flops)" << delimeter
       << "Number of GPU Blocks" << delimeter
       << "Number of Threads per Block" << delimeter
       << "Completion Status" << std::endl;

  for (VectorReduceResult result : results) {
    fout << result.deviceType << delimeter
         << result.vectorSize << delimeter
         << result.computeTimeSec << delimeter
         << result.totalTimeSec << delimeter
         << result.vectorSize / result.computeTimeSec << delimeter
         << result.vectorSize / result.totalTimeSec << delimeter
         << result.numBlocks << delimeter
         << result.threadsPerBlock << delimeter
         << result.completionStatus << std::endl;
  }
  fout.close();

  fout.clear();
  fout.open(file_path + "gpu_properties.csv");
  fout << GetGpuProperties(0) << std:: endl;
  fout.close();
}

bool
DumpResultsToRDataFile(
  const std::string& file_path,
  const std::string& file_name,
  const std::string& delimeter,
  const std::vector<VectorReduceResult>& results) {

  std::ofstream fout;
  fout.clear();
  fout.open(file_path + file_name);

  if (!fout.good()) {
    puts("Invalid output destination!");
    return false;
  }

  std::map<unsigned, double> seqeuntialTimes;
  for (VectorReduceResult result : results) {
    if (result.deviceType == CPU_DEVICE) {
      seqeuntialTimes[result.vectorSize] = result.computeTimeSec;
    }
  }

  fout << "Device Type" << delimeter
       << "Vector Size" << delimeter
       << "Runtime (s)" << delimeter
       << "Throughput (int/s)" << delimeter
       << "Speedup" << delimeter
       << "GPU Blocks" << delimeter
       << "Threads/Block" << delimeter
       << "Completion Status" << std::endl;

  for (VectorReduceResult result : results) {
    double sequentialTime = seqeuntialTimes[result.vectorSize];

    fout << result.deviceType << delimeter
         << result.method << delimeter
         << result.vectorSize << delimeter
         << result.computeTimeSec << delimeter
         << result.vectorSize / result.computeTimeSec << delimeter
         << sequentialTime / result.computeTimeSec << delimeter
         << result.numBlocks << delimeter
         << result.threadsPerBlock << delimeter
         << result.completionStatus << std::endl;

    if (result.deviceType != CPU_DEVICE) {
      fout << result.deviceType << delimeter
           << result.vectorSize << delimeter
           << result.totalTimeSec << delimeter
           << result.vectorSize / result.totalTimeSec << delimeter
           << sequentialTime / result.totalTimeSec << delimeter
           << result.numBlocks << delimeter
           << result.threadsPerBlock << delimeter
           << result.completionStatus << std::endl;
    }
  }
  fout.close();

  fout.clear();
  fout.open(file_path + "gpu_properties.csv");
  fout << GetGpuProperties(0) << std:: endl;
  fout.close();
}

std::string
GetGpuProperties(const int device_number) {
  hipDeviceProp_t cudaProperties;
  hipError_t cudaStatus = hipGetDeviceProperties(
    &cudaProperties,
    device_number
  );
  if (cudaStatus != hipSuccess) {
    char error_message[100];
    sprintf(
      error_message,
      "Device properties could for device %i not be retrieved!",
      device_number
    );
    return std::string(error_message);
  }

  char properties[2048];
  sprintf(
    properties,
    "ATTRIBUTE,VALUE\n"
    "Device Name,%s\n"
    "Cuda Version,%i.%i\n"
    "Multiprocessors,%i\n"
    "CUDA Cores,96\n"
    "Clock Rate,%i mHz\n"
    "Total Global Memory, %i MB\n"
    "Warp Size,%i\n"
    "Max Threads/Block,%i\n"
    "Max Threads-Dim,%i x %i x %i\n"
    "Max Grid Size,%i x %i x %i\n"
    "SharedMem/Block,%i KB",
    cudaProperties.name,
    cudaProperties.major,
    cudaProperties.minor,
    cudaProperties.multiProcessorCount,
    cudaProperties.clockRate / 1000,
    cudaProperties.totalGlobalMem / 1000000,
    cudaProperties.warpSize,
    cudaProperties.maxThreadsPerBlock,
    cudaProperties.maxThreadsDim[0],
    cudaProperties.maxThreadsDim[1],
    cudaProperties.maxThreadsDim[2],
    cudaProperties.maxGridSize[0],
    cudaProperties.maxGridSize[1],
    cudaProperties.maxGridSize[2],
    cudaProperties.sharedMemPerBlock / 1000
  );

  return std::string(properties);
}