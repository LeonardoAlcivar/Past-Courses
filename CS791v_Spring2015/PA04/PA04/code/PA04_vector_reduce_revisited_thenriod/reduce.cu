#include "hip/hip_runtime.h"
/**
 *
 */

#ifndef _REDUCE_CU_
#define _REDUCE_CU_ 1

#include "reduce.hpp"

#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>

#include <cstdio>


__global__
void
PlainReduceKernel(
    int* resultVector, const int* inputVector, const unsigned vectorSize) {

  extern __shared__ int sharedData[];

  const unsigned& blockTid = threadIdx.x;
  unsigned globalTid = (blockIdx.x * blockDim.x) + blockTid;
  unsigned int totalThreads = gridDim.x * blockDim.x;

  /* fetch the data from global into shared memory */
  sharedData[blockTid] = 0;
  for (unsigned i = globalTid; i < vectorSize; i += totalThreads) {
    sharedData[blockTid] += inputVector[i];
  }

  unsigned power2PaddedNumThreadsInBlock = 1;
  while (power2PaddedNumThreadsInBlock < blockDim.x) {
    power2PaddedNumThreadsInBlock *= 2;
  }
  __syncthreads();

  /* perform the reduction */
  unsigned stride = power2PaddedNumThreadsInBlock / 2;

  // unroll the first iteration so the special check to see if threads are
  // accessing valid memory with their stride can be done only once
  if (blockTid  + stride < blockDim.x && stride > 0) {
    sharedData[blockTid] += sharedData[blockTid + stride];
  }
  __syncthreads();

  // the main algorithm
  for (unsigned stride = power2PaddedNumThreadsInBlock / 4;
       stride > 0;
       stride /= 2) {

    if (blockTid < stride) {
      sharedData[blockTid] += sharedData[blockTid + stride];
    }
    __syncthreads();
  }

  // TODO: Optimize things for the last warp. Unfortunately, if we want it to
  //       be portable, it will involve a lot of if-logic that may not be
  //       worth implementing for programming and performance reasons. 

  /* store the final result */
  if (blockTid == 0) {
    resultVector[blockIdx.x] = sharedData[0];
  }
}

__global__
void
KernelInKernelReduceKernel(
    int* sum,
    int* intermediateVector,
    const int* inputVector,
    const unsigned vectorSize) {

  extern __shared__ int sharedData[];

  const unsigned& blockTid = threadIdx.x;
  unsigned globalTid = (blockIdx.x * blockDim.x) + blockTid;
  unsigned int totalThreads = gridDim.x * blockDim.x;

  if (globalTid == 0) {
    *sum = 0;
  }

  /* fetch the data from global into shared memory */
  sharedData[blockTid] = 0;
  for (unsigned i = globalTid; i < vectorSize; i += totalThreads) {
    sharedData[blockTid] += inputVector[i];
  }

  unsigned power2PaddedNumThreadsInBlock = 1;
  while (power2PaddedNumThreadsInBlock < blockDim.x) {
    power2PaddedNumThreadsInBlock *= 2;
  }
  __syncthreads();

  /* perform the reduction */
  unsigned stride = power2PaddedNumThreadsInBlock / 2;

  // unroll the first iteration so the special check to see if threads are
  // accessing valid memory with their stride can be done only once
  if (blockTid  + stride < blockDim.x && stride > 0) {
    sharedData[blockTid] += sharedData[blockTid + stride];
  }
  __syncthreads();

  // the main algorithm
  for (unsigned stride = power2PaddedNumThreadsInBlock / 4;
       stride > 0;
       stride /= 2) {

    if (blockTid < stride) {
      sharedData[blockTid] += sharedData[blockTid + stride];
    }
    __syncthreads();
  }

  // TODO: Optimize things for the last warp. Unfortunately, if we want it to
  //       be portable, it will involve a lot of if-logic that may not be
  //       worth implementing for programming and performance reasons. 

  /* store the final result */
  if (blockTid == 0) {
    intermediateVector[blockIdx.x] = sharedData[0];
    atomicAdd(sum, 1);
  }
  __threadfence();

  if (globalTid == 0) {

    while (*sum < gridDim.x) {printf("");} // a little hack to kick the block
                                           // out of the SM while synchronization
                                           // is achieved

    if (gridDim.x > (2 * 1024)) {
      KernelInKernelReduceKernel
        <<<gridDim.x / 2, blockDim.x, blockDim.x * sizeof(int)>>>(
        sum,
        intermediateVector,
        intermediateVector,
        gridDim.x
      );
    } else if (gridDim.x == 1) {
      *sum = intermediateVector[0];
    } else {
      PlainReduceKernel // TODO: more threads than elements for this call?
        <<<1, (gridDim.x / 2), (gridDim.x / 2) * sizeof(int)>>>(
        sum,
        intermediateVector,
        gridDim.x
      );
    }
  }
}

__global__
void
ThreadFenceReduceKernel(
    int* resultVector, 
    const int* inputVector,
    const unsigned vectorSize) {

 // TODO: implement for extra credit
}


__device__
void
warpReduce(volatile int* sharedData, const unsigned threadsPerBlock, const int globalTid) {
  if (threadsPerBlock >= 64) {
    sharedData[globalTid] += sharedData[globalTid + 32];
  }
  if (threadsPerBlock >= 32) {
    sharedData[globalTid] += sharedData[globalTid + 16];
  }
  if (threadsPerBlock >= 16) {
    sharedData[globalTid] += sharedData[globalTid + 8];
  }
  if (threadsPerBlock >= 8) {
    sharedData[globalTid] += sharedData[globalTid + 4];
  }
  if (threadsPerBlock >= 4) {
    sharedData[globalTid] += sharedData[globalTid + 2];
  }
  if (threadsPerBlock >= 2) {
    sharedData[globalTid] += sharedData[globalTid + 1];
  }
}

#endif //_REDUCE_CU_